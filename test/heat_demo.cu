#include "hip/hip_runtime.h"
#include <stdio.h>

#include <poisson.hpp>
#include <poisson.cuh>
#include <assertions.hpp>
#include <grid.hpp>
#include <solver.hpp>


template <typename S, typename P, typename T=double>
void convergence_test(const int num_grids, SolverOptions opts) {
        T rate = 0.0;
        T err1 = 0.0;
        T modes = 1.0;
        int l = 2;
        T h = 1.0;
        printf("MMS convergence test\n");
        {
                S tmp;
                printf("Solver: %s \n", tmp.name());
        }
        printf("Grid Size \t Iterations \t Time (ms) \t Residual \t Error \t\t Rate \n");
        for (int i = 0; i < num_grids; ++i) {
                hipEvent_t start, stop;
                hipEventCreate(&start);
                hipEventCreate(&stop);
                P problem(l, h, modes);
                S solver(problem);

                hipEventRecord(start);
                SolverOutput out = solve(solver, problem, opts);
                hipEventRecord(stop);
                hipEventSynchronize(stop);
                float elapsed = 0;
                hipEventElapsedTime(&elapsed, start, stop);

                rate = log2(err1 / out.error);
                int n = (1 << l) + 1;
                printf("%4d x %-4d \t %-7d \t %-5.5f \t %-5.5g \t %-5.5g \t %-5.5f \n", 
                       n, n,
                       out.iterations, elapsed, out.residual, out.error, rate);
                err1 = out.error;
                l++;
                h /= 2;
        }
}


int main(int argc, char **argv) {

        using Number = double;
        SolverOptions opts;
        opts.verbose = 1;
        opts.info = 10;
        opts.max_iterations = 1e4;
        opts.eps = 1e-8;
        opts.mms = 1;
        int l = 4;
        int n = (1 << l) + 1;
        double h = 1.0 / (n - 1);
        double modes = 1.0;
        using Problem = Poisson<Number>;

        {
                using CUDAProblem = CUDAPoisson<L1NORM, Number>;
                using CUDASmoother = CUDAGaussSeidelRedBlack;
                using CUDAMG = CUDAMultigrid<CUDASmoother, CUDAProblem, Number>;
                
                CUDAProblem problem(l, h, modes);
                CUDAMG solver(problem);

                hipEvent_t start, stop;
                hipEventCreate(&start);
                hipEventCreate(&stop);
                hipEventRecord(start);
                SolverOutput out = solve(solver, problem, opts);
                hipEventRecord(stop);
                hipEventSynchronize(stop);
                float elapsed = 0;
                hipEventElapsedTime(&elapsed, start, stop);
                printf("Iterations: %d, Residual: %g, Time(ms): %g \n", out.iterations, out.residual, elapsed);
        }

}
